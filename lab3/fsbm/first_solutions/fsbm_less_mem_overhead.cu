/************************************************************************/
/* File: interPrediction.c                                              */
/* Author: Nuno Roma <Nuno.Roma@tecnico.ulisboa.pt                      */
/* Date: February 23th, 2024                                            */
/************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define FSBM 0 // Full-Search Block Matching (FSBM) motion estimation algorithm
#define SS 1   // (Three/Four) Step-Search (SS) block Matching motion estimation algorithm
#define TZS 2  // Test Zonal Search (TZS) block matching motion estimation algorithm

#define SEARCH_RANGE 64 // Search range (at each direction)
#define BLOCK_SIZE 32   // Block size (at each direction)
#define iRASTER 5       // TZS iRaster parameter

#define BigSAD 999999 // it could be any other big integer...

typedef struct
{
    char *video_name; // YUV input file
    int width;        // luminance width
    int height;       // luminance height
    int frames;       // number of frames to process
    int algorithm;    // motion estimation algorithm
    int searchRange;  // search range (at each direction)
    int blockSize;    // block size (at each direction)
    int iRaster;      // TZS iRaster parameter
    int debug;        // verbose mode
} Parameters;

typedef struct
{
    int vec_x;
    int vec_y;
    int sad;
    int bestDist;
} BestResult;

/************************************************************************************/
void getLumaFrame(int *frame_mem, FILE *yuv_file, Parameters p)
{
    int count;
    for (int r = 0; r < p.height; r++)
        for (int c = 0; c < p.width; c++)
            count = fread(&(frame_mem[r * p.width + c]), 1, 1, yuv_file);
    count++; // avoid warning

    // Skips the color Cb and Cr components in the YUV 4:2:0 file
    fseek(yuv_file, p.width * p.height / 2, SEEK_CUR);
}
/************************************************************************************/
void setLumaFrame(int **frame_mem, FILE *yuv_file, Parameters p)
{
    __uint8_t temp;
    for (int r = 0; r < p.height; r++)
        for (int c = 0; c < p.width; c++)
        {
            temp = (__uint8_t)frame_mem[r][c];
            fwrite(&temp, 1, 1, yuv_file);
        }
    // writes 2*(height/2)*(width/2) values to fill in chrominance part with 128
    temp = (__uint8_t)128;
    for (int r = 0; r < p.height / 2; r++)
        for (int c = 0; c < p.width; c++)
        {
            fwrite(&temp, 1, 1, yuv_file);
        }
}
/************************************************************************************/
void reconstruct(int **rec_frame, int *ref_frame, int i, int j, Parameters p, BestResult *MV)
{
    for (int a = i; a < i + p.blockSize; a++)
        for (int b = j; b < j + p.blockSize; b++)
            if ((0 <= a + MV->vec_x) && (a + MV->vec_x < p.height) && (0 <= b + MV->vec_y) && (b + MV->vec_y < p.width))
                rec_frame[a][b] = ref_frame[(a + MV->vec_x) * (p.width) + b + MV->vec_y];
}
/************************************************************************************/
unsigned long long computeResidue(int **res_frame, int *curr_frame, int **rec_frame, Parameters p)
{
    unsigned long long accumulatedDifference = 0;
    int difference;
    for (int a = 0; a < p.height; a++)
        for (int b = 0; b < p.width; b++)
        {
            difference = curr_frame[a * p.width + b] - rec_frame[a][b];
            if (difference < 0)
                difference = -difference;
            if (255 < difference)
                difference = 255;
            res_frame[a][b] = difference;
            accumulatedDifference += difference;
        }
    return (accumulatedDifference);
}
/************************************************************************************/
void getBlock(int *block, int *frame, int i, int j, Parameters p)
{
    for (int m = 0; m < p.blockSize; m++)
        for (int n = 0; n < p.blockSize; n++)
            block[m * p.blockSize + n] = frame[(i + m) * p.width + j + n];
}
/************************************************************************************/
void getSearchArea(int *searchArea, int *frame, int i, int j, Parameters p)
{
    for (int m = -p.searchRange; m < p.searchRange + p.blockSize; m++)
        for (int n = -p.searchRange; n < p.searchRange + p.blockSize; n++)
            if (((0 <= (i + m)) && ((i + m) < p.height)) && ((0 <= (j + n)) && ((j + n) < p.width)))
                searchArea[(p.searchRange + m) * (2 * p.searchRange + p.blockSize) + (p.searchRange + n)] = frame[(i + m) * p.width + j + n];
            else
                searchArea[(p.searchRange + m) * (2 * p.searchRange + p.blockSize) + (p.searchRange + n)] = 0;
}
/************************************************************************************/

__global__ void sum_SAD(int *curr, int *search, int *d_results, int posX, int posY, int step_search)
{
    unsigned int column = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y;
    unsigned int column_search = column + posY;
    unsigned int row_search = row + posX;
    unsigned int tid = threadIdx.x;
    __shared__ int sad[BLOCK_SIZE];
    sad[tid] = abs(curr[row * blockDim.x + column] - search[row_search * step_search + column_search]);
    __syncthreads();
    for (int i = blockDim.x >> 1; i > 0; i = i >> 1)
    {
        if (tid < i)
        {
            sad[tid] += sad[tid + i];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        d_results[blockIdx.y] = sad[0];
    }
}

__global__ void sum_results(int *d_results)
{
    unsigned int tid = threadIdx.x;
    __shared__ int sad[BLOCK_SIZE];
    sad[tid] = d_results[tid];
    __syncthreads();
    for (int i = blockDim.x >> 1; i > 0; i = i >> 1)
    {
        if (tid < i)
        {
            sad[tid] += sad[tid + i];
        }
        __syncthreads();
    }
    if (tid == 0)
    {
        d_results[blockIdx.y] = sad[0];
    }
}
void SAD(BestResult *bestResult, int *d_CurrentBlock, int *d_SearchArea, int rowIdx, int colIdx, int k, int m, Parameters p, int *d_results)
{
    // k, m: displacement (motion vector) under analysis (in the search area)

    int sad[1] = {0};
    int posX = p.searchRange + k; // normalized coordinates within search area, between 0 and 2*searchRange
    int posY = p.searchRange + m; // normalized coordinates within search area, between 0 and 2*searchRange
    // checks if search area range is valid (inside frame borders) and if current block range is valid (inside frame borders)
    if ((-p.searchRange <= k) && (k <= p.searchRange) &&
        (-p.searchRange <= m) && (m <= p.searchRange) &&
        (0 <= (rowIdx + posX)) && ((rowIdx + posX) < p.height) &&
        (0 <= (colIdx + posY)) && ((colIdx + posY) < p.width))
    {
        // computes SAD disparity, by comparing the current block with the reference block at (k,m)
        int tamanhoy = BLOCK_SIZE;        // SE mudar alterar o tamanho do block results na função fullSearch
        dim3 gridDist(1, p.blockSize, 1); // X -> p.blockSize (threadIdx.x)
        dim3 blockDist(tamanhoy, 1, 1);   // Y    -> tamanhoy (blockIdx.y)

        sum_SAD<<<gridDist, blockDist>>>(d_CurrentBlock, d_SearchArea, d_results, posX, posY, (2 * p.searchRange + p.blockSize));
        sum_results<<<1, tamanhoy>>>(d_results);
        if (hipMemcpy(sad, d_results, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
        {
            printf("FAILED TO COPY results DATA TO THE host\n");
            exit(0);
        }

        // for (int i = 0; i < tamanhoy; i++)
        // {
        //     sad += results[i];
        // }
        // printf("GPU -> %d\n", sad);
        // compares the obtained sad with the best so far for that block
        if (sad[0] < bestResult->sad)
        {
            bestResult->sad = sad[0];
            bestResult->vec_x = k;
            bestResult->vec_y = m;
        }
    }
}

void fullSearch(BestResult *bestResult, int *CurrentBlock, int *SearchArea, int rowIdx, int colIdx, Parameters p)
{
    bestResult->sad = BigSAD;
    bestResult->bestDist = 0;
    bestResult->vec_x = 0;
    bestResult->vec_y = 0;
    int *d_CurrentBlock, *d_SearchArea;
    int SizeInBytes_curr = p.blockSize * p.blockSize * sizeof(int);
    int SizeInBytes_search = (2 * p.searchRange + p.blockSize) * (2 * p.searchRange + p.blockSize) * sizeof(int);
    int *d_results;
    if (hipMalloc((void **)&d_CurrentBlock, SizeInBytes_curr) != hipSuccess)
    {
        printf("CANNOT ALLOCATE d_CurrentBlock");
        exit(0);
    }
    if (hipMalloc((void **)&d_SearchArea, SizeInBytes_search) != hipSuccess)
    {
        printf("CANNOT ALLOCATE d_CurrentBlock");
        exit(0);
    }
    if (hipMemcpy(d_CurrentBlock, CurrentBlock, SizeInBytes_curr, hipMemcpyHostToDevice) != hipSuccess)
    {
        printf("FAILED TO COPY CurrentBlock DATA TO THE DEVICE\n");
        exit(0);
    }
    if (hipMemcpy(d_SearchArea, SearchArea, SizeInBytes_search, hipMemcpyHostToDevice) != hipSuccess)
    {
        printf("FAILED TO COPY SearchArea DATA TO THE DEVICE\n");
        exit(0);
    }

    if (hipMalloc((void **)&d_results, BLOCK_SIZE * sizeof(int)) != hipSuccess)
    {
        printf("CANNOT ALLOCATE d_results");
        exit(0);
    }

    for (int iStartX = -p.searchRange; iStartX < p.searchRange; iStartX++)
    {
        for (int iStartY = -p.searchRange; iStartY < p.searchRange; iStartY++)
        {
            SAD(bestResult, d_CurrentBlock, d_SearchArea, rowIdx, colIdx, iStartX, iStartY, p, d_results);
        }
    }
    hipFree(d_CurrentBlock);
    hipFree(d_SearchArea);
    hipFree(d_results);
}
/************************************************************************************/
void MotionEstimation(BestResult **motionVectors, int *curr_frame, int *ref_frame, Parameters p)
{
    BestResult *bestResult;

    int *CurrentBlock = (int *)malloc(p.blockSize * p.blockSize * sizeof(int));

    int *SearchArea = (int *)malloc((2 * p.searchRange + p.blockSize) * (2 * p.searchRange + p.blockSize) * sizeof(int));

    for (int rowIdx = 0; rowIdx < (p.height - p.blockSize + 1); rowIdx += p.blockSize)
        for (int colIdx = 0; colIdx < (p.width - p.blockSize + 1); colIdx += p.blockSize)
        {
            // Gets current block and search area data
            getBlock(CurrentBlock, curr_frame, rowIdx, colIdx, p);
            getSearchArea(SearchArea, ref_frame, rowIdx, colIdx, p);
            bestResult = &(motionVectors[rowIdx / p.blockSize][colIdx / p.blockSize]);
            // Runs the motion estimation algorithm on this block
            switch (p.algorithm)
            {
            case FSBM:
                fullSearch(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, p);
                break;
            // case TZS:
            //     TZSearch(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, p);
            //     break;
            // case SS:
            //     StepSearch(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, p);
            //     break;
            default:
                break;
            }
        }
    free(CurrentBlock);
    free(SearchArea);
}

/************************************************************************************/
int main(int argc, char **argv)
{

    struct timespec t0, t1;
    unsigned long long accumulatedResidue = 0;

    // Read input parameters
    if (argc != 7)
    {
        printf("USAGE: %s <videoPath> <Width> <Height> <NFrames> <ME Algorithm: 0=FSBM; 1=SS; 2=TZS> <Debug Mode: 0=silent; 1=verbose>\n", argv[0]);
        exit(1);
    }
    Parameters p;
    p.video_name = argv[1];
    p.width = atoi(argv[2]);
    p.height = atoi(argv[3]);
    p.frames = atoi(argv[4]);
    p.algorithm = atoi(argv[5]);
    p.searchRange = SEARCH_RANGE; // Search range (at each direction)
    p.blockSize = BLOCK_SIZE;     // Block size (at each direction)
    p.iRaster = iRASTER;          // TZS iRaster parameter
    p.debug = atoi(argv[6]);

    switch (p.algorithm)
    {
    case FSBM:
        printf("Running FSBM algorithm\n");
        break;
    case SS:
        printf("Running Step-Search algorithm\n");
        break;
    case TZS:
        printf("Running TZS algorithm\n");
        break;
    default:
        printf("ERROR: Invalid motion estimation algorithm\n");
        exit(-1);
    }

    // Video files
    FILE *video_in;
    FILE *residue_out;
    FILE *reconst_out;
    video_in = fopen(p.video_name, "rb");
    residue_out = fopen("residue.yuv", "wb");
    reconst_out = fopen("reconst.yuv", "wb");
    if (!video_in || !residue_out || !reconst_out)
    {
        printf("Opening input/output file error\n");
        exit(1);
    }

    // Frame memory allocation
    int *curr_frame = (int *)malloc(p.width * p.height * sizeof(int *));
    int *ref_frame = (int *)malloc(p.width * p.height * sizeof(int *));
    int **res_frame = (int **)malloc(p.height * sizeof(int *));
    int **rec_frame = (int **)malloc(p.height * sizeof(int *));
    for (int i = 0; i < p.height; i++)
    {
        res_frame[i] = (int *)malloc(p.width * sizeof(int));
        rec_frame[i] = (int *)malloc(p.width * sizeof(int));
    }

    // Memory allocation of result table
    BestResult **motionVectors = (BestResult **)malloc(p.height / p.blockSize * sizeof(BestResult *));
    for (int i = 0; i < p.height / p.blockSize; i++)
        motionVectors[i] = (BestResult *)malloc(p.width / p.blockSize * sizeof(BestResult));
    BestResult *MV;

    clock_gettime(CLOCK_REALTIME, &t0);
    // Read first frame
    getLumaFrame(curr_frame, video_in, p); // curr_frame contains the current luminance frame
    //
    // int *d_curr_frame, *d_ref_frame;
    // int SizeInBytes = p.width * p.height * sizeof(int);
    // if (cudaMalloc((void **)&d_curr_frame, SizeInBytes) != cudaSuccess)
    // {
    //     printf("CANNOT ALLOCATE d_curr_frame");
    //     exit(0);
    // }
    // if (cudaMalloc((void **)&d_ref_frame, SizeInBytes) != cudaSuccess)
    // {
    //     printf("CANNOT ALLOCATE d_ref_frame");
    //     exit(0);
    // }

    for (int frameNum = 0; frameNum < p.frames; frameNum++)
    {
        int *temp;
        temp = ref_frame;
        ref_frame = curr_frame; // ref_frame contains the previous (reference) luminance frame
        curr_frame = temp;
        // cudaMemcpy()
        getLumaFrame(curr_frame, video_in, p); // curr_frame contains the current luminance frame

        // Process the current frame, one block at a time, to obatin an array with the motion vectors and SAD values

        MotionEstimation(motionVectors, curr_frame, ref_frame, p);

        // Recustruct the predicted frame using the obtained motion vectors
        for (int rowIdx = 0; rowIdx < p.height - p.blockSize + 1; rowIdx += p.blockSize)
        {
            for (int colIdx = 0; colIdx < p.width - p.blockSize + 1; colIdx += p.blockSize)
            {
                // Gets best candidate block information
                MV = &(motionVectors[rowIdx / p.blockSize][colIdx / p.blockSize]);

                // Reconstructs current block using  the obtained motion estimation information
                reconstruct(rec_frame, ref_frame, rowIdx, colIdx, p, MV);

                // Print vector information
                if (p.debug)
                    printf("Frame %d : Block [%4d , %4d] = (%3d,%3d), SAD= %d\n", frameNum, colIdx, rowIdx, MV->vec_y, MV->vec_x, MV->sad);
            }
        }
        // Reconstructs borders of the frame not convered by motion estimation
        for (int r = 0; r < p.height; r++)
            for (int c = 0; c < p.width; c++)
                if (r > (p.height - p.blockSize + 1) || c > (p.width - p.blockSize + 1))
                    rec_frame[r][c] = ref_frame[r * p.width + c];

        // Compute residue block
        accumulatedResidue += computeResidue(res_frame, curr_frame, rec_frame, p);

        // Save reconstructed and residue frames
        setLumaFrame(rec_frame, reconst_out, p);
        setLumaFrame(res_frame, residue_out, p);
    }
    clock_gettime(CLOCK_REALTIME, &t1);
    printf("%lf seconds elapsed \n", (t1.tv_sec - t0.tv_sec) + (t1.tv_nsec - t0.tv_nsec) * 1e-9);
    printf("Accumulated Residue = %llu \n", accumulatedResidue);

    // Frame memory free
    for (int i = 0; i < p.height; i++)
    {
        free(res_frame[i]);
        free(rec_frame[i]);
    }
    free(curr_frame);
    free(ref_frame);
    free(res_frame);
    free(rec_frame);
    return 0;
}

// 190 mais ou menos, por baixo do SAD
/************************************************************************************/
// void StepSearch(BestResult *bestResult, int **CurrentBlock, int **SearchArea, int rowIdx, int colIdx, Parameters p)
// {

//     bestResult->sad = BigSAD;
//     bestResult->vec_x = 0;
//     bestResult->vec_y = 0;

//     // First prediction, at the center of the search area
//     int CenterX = 0;
//     int CenterY = 0;
//     // SAD(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, CenterX, CenterY, p);

//     // Furthest search center
//     int Distance = (p.searchRange) >> 1; // Initial distance = search range/2
//     while (Distance >= 1)
//     {
//         // SAD(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, CenterX - Distance, CenterY - Distance, p); // Top-Left
//         // SAD(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, CenterX - Distance, CenterY + 0, p);        // Top-Center
//         // SAD(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, CenterX - Distance, CenterY + Distance, p); // Top-Right
//         // SAD(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, CenterX + 0, CenterY - Distance, p);        // Center-Left
//         // SAD(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, CenterX + 0, CenterY + Distance, p);        // Center-Right
//         // SAD(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, CenterX + Distance, CenterY - Distance, p); // Top-Left
//         // SAD(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, CenterX + Distance, CenterY + 0, p);        // Top-Center
//         // SAD(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, CenterX + Distance, CenterY + Distance, p); // Top-Right
//         //  At this point, (bestResult->vec_x,bestResult->vec_y) marks the best search point and will be considered as the next search center
//         CenterX = bestResult->vec_x;
//         CenterY = bestResult->vec_y;
//         // Divides the search distance by 2
//         Distance >>= 1;
//     }
// }
// /************************************************************************************/
// void xTZ8PointDiamondSearch(BestResult *bestResult, int **CurrentBlock, int **SearchArea, int rowIdx, int colIdx, int centroX, int centroY, int iDist, Parameters p)
// {
//     BestResult localBest;
//     localBest.sad = bestResult->sad;
//     localBest.bestDist = iDist;
//     localBest.vec_x = 0;
//     localBest.vec_y = 0;

//     if (iDist == 1)
//     {
//         // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, centroX - iDist, centroY, p);
//         // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, centroX, centroY - iDist, p);
//         // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, centroX, centroY + iDist, p);
//         // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, centroX + iDist, centroY, p);
//     }
//     else
//     {
//         int iTop = centroY - iDist;
//         int iBottom = centroY + iDist;
//         int iLeft = centroX - iDist;
//         int iRight = centroX + iDist;
//         if (iDist <= 8)
//         {
//             int iTop_2 = centroY - (iDist >> 1);
//             int iBottom_2 = centroY + (iDist >> 1);
//             int iLeft_2 = centroX - (iDist >> 1);
//             int iRight_2 = centroX + (iDist >> 1);
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, centroX, iTop, p);
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iLeft, centroY, p);
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iRight, centroY, p);
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, centroX, iBottom, p);
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iLeft_2, iTop_2, p);
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iRight_2, iTop_2, p);
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iLeft_2, iBottom_2, p);
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iRight_2, iBottom_2, p);
//         }
//         else
//         {
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, centroX, iTop, p);
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iLeft, centroY, p);
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iRight, centroY, p);
//             // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, centroX, iBottom, p);
//             for (int index = 1; index < 4; index++)
//             {
//                 int iPosYT = iTop + ((iDist >> 2) * index);
//                 int iPosYB = iBottom - ((iDist >> 2) * index);
//                 int iPosXL = centroX - ((iDist >> 2) * index);
//                 int iPosXR = centroX + ((iDist >> 2) * index);
//                 // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iPosXL, iPosYT, p);
//                 // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iPosXR, iPosYT, p);
//                 // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iPosXL, iPosYB, p);
//                 // SAD(&(localBest), CurrentBlock, SearchArea, rowIdx, colIdx, iPosXR, iPosYB, p);
//             }
//         }
//     }
//     if (localBest.sad < bestResult->sad)
//     {
//         bestResult->sad = localBest.sad;
//         bestResult->bestDist = localBest.bestDist;
//         bestResult->vec_x = localBest.vec_x;
//         bestResult->vec_y = localBest.vec_y;
//     }
// }
// /************************************************************************************/
// void TZSearch(BestResult *bestResult, int **CurrentBlock, int **SearchArea, int rowIdx, int colIdx, Parameters p)
// {
//     int bestX, bestY;
//     bestResult->sad = BigSAD;
//     bestResult->bestDist = 0;
//     bestResult->vec_x = 0;
//     bestResult->vec_y = 0;

//     // First prediction, at the center of the search area
//     // SAD(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, 0, 0, p);

//     // Initial Search: iDist in [1, 2, 4, 8, 16, 32, 64]
//     int iDist = 1;
//     while (iDist <= p.searchRange)
//     {
//         xTZ8PointDiamondSearch(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, 0, 0, iDist, p);
//         iDist <<= 1;
//     }

//     // Raster Search
//     bestX = bestResult->vec_x;
//     bestY = bestResult->vec_y;
//     if ((bestX > p.iRaster) || (bestY > p.iRaster) || (-bestX > p.iRaster) || (-bestY > p.iRaster))
//     {
//         int Top = -(int)(p.searchRange / 2);
//         int Bottom = (int)(p.searchRange / 2);
//         int Left = -(int)(p.searchRange / 2);
//         int Right = (int)(p.searchRange / 2);
//         for (int iStartY = Top; iStartY < Bottom; iStartY += p.iRaster)
//             for (int iStartX = Left; iStartX < Right; iStartX += p.iRaster)
//                 // SAD(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, iStartX, iStartY, p);
//                 printf("Ahhhhhh");
//     }

//     // Refinement
//     bestX = bestResult->vec_x;
//     bestY = bestResult->vec_y;
//     int RefinementCount = 0;
//     if ((bestX != 0) || (bestY != 0))
//         while ((bestResult->vec_x == bestX) && (bestResult->vec_y == bestY))
//         {
//             iDist = 1;
//             while (iDist <= p.searchRange)
//             {
//                 xTZ8PointDiamondSearch(bestResult, CurrentBlock, SearchArea, rowIdx, colIdx, bestX, bestY, iDist, p);

//                 if (((4 <= iDist) && (bestResult->bestDist == 0)) ||
//                     ((8 <= iDist) && (bestResult->bestDist <= 1)) ||
//                     ((16 <= iDist) && (bestResult->bestDist <= 2)) ||
//                     ((32 <= iDist) && (bestResult->bestDist <= 4)))
//                     break;

//                 iDist <<= 1;
//             }
//             if (((bestResult->vec_x == bestX) && (bestResult->vec_y == bestY)) || (RefinementCount == 7))
//                 break;
//             else
//             {
//                 bestX = bestResult->vec_x;
//                 bestY = bestResult->vec_y;
//                 RefinementCount += 1;
//             }
//         }
// }
// /************************************************************************************/